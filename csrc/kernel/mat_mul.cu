#include "hip/hip_runtime.h"
#include "./configs.cuh"

namespace Kernel {
namespace Matmul {

// (N, N) * (N, N)
__global__ void matMulNaive(const float* M, const float* N, float* P, const int Width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < Width) && (col < Width)) {
		float sum = 0.0f;
		for (int k = 0; k < Width; ++k) {
			sum += M[row * Width + k] * N[k * Width + col];
		}
		P[row * Width + col] = sum;
	}
}

#define TILE_WIDTH 16
__global__ void matMulTile(const float *M, const float *N, float *P, const int Width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // int bx
}

void launch_mat_mm(const float *M, const float *N, float *P, const int Width) {
  SETUP_KERNEL_CONFIG(16, 16, Width, Width);
  LAUNCH_KERNEL(matMulNaive, M, N, P, Width);
}

} //namespace Matmul
} // namespace Kernel

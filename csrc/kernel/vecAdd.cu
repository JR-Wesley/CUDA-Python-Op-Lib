#include "hip/hip_runtime.h"
#include "./configs.cuh"

namespace Ops {

namespace Elemwise {

// run on CPU and GPU
__host__ __device__ __forceinline__ float add(float a, float b) {
	return a + b;
}

// Compute vector sum C_h = A_h + B_h
void vecAddHost(const float* A_h, const float* B_h, float* C_h, const int N) {
	for (int i = 0; i < N; ++i) {
		C_h[i] = add(A_h[i], B_h[i]);
	}
}

__global__ void vecAddKernel(const float* A, const float* B, float* C, const unsigned int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		C[i] = add(A[i], B[i]);
	}
}

__device__ __forceinline__ float4 getFloat4(float* ptr, const int idx) {
    return *reinterpret_cast<float4*>(&ptr[idx]);
}

__global__ void vecAddKernelFloat4(float* A, float* B, float* C, const unsigned int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = i * 4;

  if (idx < N) {
    float4 a = getFloat4(A, idx);
    float4 b = getFloat4(B, idx);
    float4 c = make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);

    if (idx + 0 < N) C[idx + 0] = c.x;
    if (idx + 1 < N) C[idx + 1] = c.y;
    if (idx + 2 < N) C[idx + 2] = c.z;
    if (idx + 3 < N) C[idx + 3] = c.w;
  }
}

void vecAddDevice(const float* A_h, const float* B_h, float* C_h, const int N, const bool if_float4=false) {
	int size = N * sizeof(float);
	float* A_d, *B_d, *C_d;

	// Allocate GPU memory
	hipMalloc((void**) &A_d, size);
	hipMalloc((void**) &B_d, size);
	hipMalloc((void**) &C_d, size);

	// Copy to the GPU
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	// launch the kernel function (a grid of threads)
	const unsigned int numThreadsPerBlock = 512;
  unsigned int numBlocks;
	// ceil(N/512)
  if (if_float4) {
    numBlocks = (N + 4 * numThreadsPerBlock - 1) / (4 * numThreadsPerBlock);
  } else {
    numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
  }
	vecAddKernel<<<numBlocks, numThreadsPerBlock>>>(A_d, B_d, C_d, N);

	// Copy to the CPU
	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

	// Deallocate GPU memory
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}

void launch_vec_add(const float* A, const float* B, float* C, unsigned int N) {
  const int blockSize = 256;
  const int gridSize = (N + blockSize - 1) / blockSize;

  vecAddKernel<<<gridSize, blockSize>>>(A, B, C, N);
}

} // namespace Elemwise
} // namespace Ops

#include "hip/hip_runtime.h"
#include "./configs.cuh"
 // dim3 block_size(BLOCK_SIZE);  // BLOCK_SIZE 是通过宏定义的某个数字
 // dim3 grid_size(CIEL(N, BLOCK_SIZE));
 // reduce_v1<<<grid_size, block_size>>>(d_x, d_y, N);

namespace Reduce {

__global__ void sum_naive(const float* input, float* output, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N) {
    atomicAdd(output, input[idx]);
  }
}

// __global__ void sum_shmem(const float* input, float* output, int N) {
//     int tid = threadIdx.x;
//     int idx = blockDim.x * blockIdx.x + threadIdx.x;
//     __shared__ float input_s[BLOCK_SIZE];
//
//     // 1. 搬运和线程数量(blockDim.x)相等的数据，到当前block的共享内存中
//     input_s[tid] = (idx < N) ? input[idx] : 0.0f;
//     __syncthreads();
//
//     // 2. 用1/2, 1/4, 1/8...的线程进行折半归约
//     for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
//         if (tid < offset) {  // 2.折半归约
//             input_s[tid] += input_s[tid + offset];
//         }
//         __syncthreads();
//     }
//
//     // 3. 每个block的第一个线程将计算结果累加到输出中
//     if (tid == 0) atomicAdd(output, input_s[0]);
// }
//
//  __global__ void reduce_v3(float* d_x, float* d_y, const int N) {
//      __shared__ float s_y[32];  // 仅需要32个，因为一个block最多1024个线程，最多1024/32=32个warp
//
//      int idx = blockDim.x * blockIdx.x + threadIdx.x;
//      int warpId = threadIdx.x / warpSize;  // 当前线程属于哪个warp
//      int laneId = threadIdx.x % warpSize;  // 当前线程是warp中的第几个线程
//
//      float val = (idx < N) ? d_x[idx] : 0.0f;  // 搬运d_x[idx]到当前线程的寄存器中
//      #pragma unroll
//      for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
//          val += __shfl_down_sync(0xFFFFFFFF, val, offset);   // 在一个warp里折半归约
//      }
//
//      if (laneId == 0) s_y[warpId] = val;  // 每个warp里的第一个线程，负责将数据存储到shared mem中
//      __syncthreads();
//
//      if (warpId == 0) {  // 使用每个block中的第一个warp对s_y进行最后的归约
//          int warpNum = blockDim.x / warpSize;  // 每个block中的warp数量
//          val = (laneId < warpNum) ? s_y[laneId] : 0.0f;
//          for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
//              val += __shfl_down_sync(0xFFFFFFFF, val, offset);
//          }
//          if (laneId == 0) atomicAdd(d_y, val);  // 使用此warp中的第一个线程，将结果累加到输出
//      }
//  }

}


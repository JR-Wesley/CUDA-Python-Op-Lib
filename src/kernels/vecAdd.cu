#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>

// run on CPU and GPU
__host__ __device__ float add(float a, float b) {
	return a + b;
}

// Compute vector sum C_h = A_h + B_h
void vecAddHost(float* A_h, float* B_h, float* C_h, int N) {
	for (int i = 0; i < N; ++i) {
		C_h[i] = add(A_h[i], B_h[i]);
	}
}

__global__ void vecAddKernel(float* A, float* B, float* C, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		C[i] = add(A[i], B[i]);
	}
}

void vecAddDevice(float* A_h, float* B_h, float* C_h, int N) {
	int size = N * sizeof(float);
	float* A_d, *B_d, *C_d;

	// Allocate GPU memory
	hipMalloc((void**) &A_d, size);
	hipMalloc((void**) &B_d, size);
	hipMalloc((void**) &C_d, size);

	// Copy to the GPU
	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	// launch the kernel function (a grid of threads)
	const unsigned int numThreadsPerBlock = 512;
	// ceil(N/512)
	const unsigned int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
	vecAddKernel<<<numBlocks, numThreadsPerBlock>>>(A_d, B_d, C_d, N);

	// Copy to the CPU
	hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

	// Deallocate GPU memory
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);
}


int main(int argc, char** argv) {
	
	hipDeviceSynchronize();

    // Allocate host memory
	int N = (1 << 25);
    float* A = (float*)malloc(N * sizeof(float));
    float* B = (float*)malloc(N * sizeof(float));
    float* C = (float*)malloc(N * sizeof(float));
	for (unsigned i = 0; i < N; ++i) {
		A[i] = rand();
		B[i] = rand();
	}

	clock_t start_host = clock();
	vecAddHost(A, B, C, N);
	clock_t end_host = clock();
	double time_host = (double)(end_host - start_host) / CLOCKS_PER_SEC;

	clock_t start_device = clock();
	vecAddDevice(A, B, C, N);
	clock_t end_device = clock();
	double time_device = (double)(end_device - start_device) / CLOCKS_PER_SEC;

	printf("Time taken for host: %f seconds\n", time_host);
	printf("Time taken for device: %f seconds", time_device);

	return 0;
}

#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" void launch_gemm_kernel(
    const float* A,
    const float* B,
    float* C,
    int M, int N, int K,
    hipStream_t stream
) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);

    const float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        B, N,     // 注意：PyTorch 是 row-major，cublas 是 column-major
        A, K,
        &beta,
        C, N
    );

    hipblasDestroy(handle);
}

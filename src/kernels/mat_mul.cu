#include <__clang_cuda_builtin_vars.h>
#include <hip/hip_runtime.h>

// (N, N) * (N, N)
__global__ void matMulKernel(const float* A, const float* B, float* C, unsigned int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < N) && (col < N)) {
		float sum = 0.0f;
		for (int i = 0; i < N; ++i) {
			sum += A[row * N + i] * B[i * N + col];
		}
		C[row * N + col] = sum;
	}
}


#include <hip/hip_runtime.h>


__global__ void rgb2gray_kernel ( unsigned char* r, unsigned char* g, unsigned char* b, unsigned char* gray, unsigned int width, unsigned int height) {
	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

	if ( row < height && col < width) {
		unsigned int i = row*width + col;
		gray[i] = r[i]*3/10 + g[i]*6/10 + b[i]*1/10;
	}
}


int main() {
// const unsigned int width = 32;
// const unsigned int height = 32;

}

